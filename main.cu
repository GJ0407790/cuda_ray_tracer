#include <iostream>

#include "config.hpp"
#include "config_utils.cuh"
#include "draw.cuh"
#include "parse.hpp"
#include "libpng.h"

using std::cout;
using std::endl;

#define CUDA_CHECK(call)                                                   \
    {                                                                      \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                          \
            std::cerr << "CUDA Error in " << __FILE__ << " at line "       \
                      << __LINE__ << " : " << hipGetErrorString(err)      \
                      << std::endl;                                        \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

int main(int argc, char* argv[]){
	if(argc != 2)
	{
		std::cerr << "Use case: make run file=your/file.txt" << endl;
		exit(1);
	}

	StlConfig host_stl_config;
	//parse the inputs into host config
	parseInput(argv, host_stl_config);
	std::cout << "After parseInput" << std::endl;

	RawConfig host_raw_config;
	// init from stl config
	initRawConfigFromStl(host_stl_config, host_raw_config);
	std::cout << "After initRawConfigFromStl" << std::endl;
	
	// device allocations
	copyRawConfigToDevice(host_raw_config);
	CUDA_CHECK(hipPeekAtLastError());
	std::cout << "After copyRawConfigToDevice" << std::endl;

	// copy config to gpu
	RawConfig* d_raw_config;
	CUDA_CHECK(hipMalloc(&d_raw_config, sizeof(RawConfig)));
	CUDA_CHECK(hipMemcpy(d_raw_config, &host_raw_config, sizeof(RawConfig), hipMemcpyHostToDevice));

	// create the rgba array in gpu
	RGBA* d_image;
	CUDA_CHECK(hipMalloc(&d_image, host_stl_config.width * host_stl_config.height * sizeof(RGBA)));

	render(d_image, host_stl_config.width, host_stl_config.height, host_stl_config.aa, d_raw_config);
	CUDA_CHECK(hipDeviceSynchronize());
	std::cout << "After render" << std::endl;

	// create the image
	Image img(host_stl_config.width, host_stl_config.height);
	// copy the rendered image from gpu to cpu
	CUDA_CHECK(hipMemcpy(img[0], d_image, host_stl_config.width * host_stl_config.height * sizeof(RGBA), hipMemcpyDeviceToHost));

	std::string output_path = host_stl_config.filename;
	img.save(output_path.c_str());

	// free host memory
	freeStlConfig(host_stl_config);

	// // free gpu memory
	CUDA_CHECK(hipFree(d_image));
	CUDA_CHECK(hipFree(d_raw_config));
	// freeRawConfigDeviceMemory(host_raw_config);
	// CUDA_CHECK(hipPeekAtLastError());
}
